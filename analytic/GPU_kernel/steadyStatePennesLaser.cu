
#include <hip/hip_runtime.h>
/*
 * Example Matlab cuda kernel interface.
 */


__device__
void pointSource(double rVar, double r1, double r2, double wPerf, double cblood, double kCond, double mueff, double u0, double ua, double Power, double *temperature )
{

   double pi = 3.141592653589793;
//*temperature  =ua-(exp(-mueff*rVar)*powf(mueff,2)*Power)/(4*kCond*powf(mueff,2)*pi*rVar-4*cblood*pi*rVar*wPerf)+(exp(-mueff*(r1+rVar)+(r1+r2-rVar)*sqrt((cblood*wPerf)/kCond))*(exp(r1*(mueff+sqrt((cblood*wPerf)/kCond)))*powf(mueff,2)*Power*powf(r2,2)*(1+mueff*rVar);

*temperature=ua-(exp(-mueff*rVar)*powf(mueff,2)*Power)/(4*kCond*powf(mueff,2)*pi*rVar-4*cblood*pi*rVar*wPerf)+(exp(-mueff*(r1+rVar)+(r1+r2-rVar)*sqrt((cblood*wPerf)/kCond))*(exp(r1*(mueff+sqrt((cblood*wPerf)/kCond)))*powf(mueff,2)*Power*powf(r2,2)*(1+mueff*rVar)-exp(mueff*rVar+r2*sqrt((cblood*wPerf)/kCond))*powf(mueff,2)*Power*powf(rVar,2)*(-1+r2*sqrt((cblood*wPerf)/kCond))+4*exp(mueff*(r1+rVar)+r2*sqrt((cblood*wPerf)/kCond))*pi*r1*powf(rVar,2)*(u0-ua)*(-kCond*powf(mueff,2)+cblood*wPerf)*(-1+r2*sqrt((cblood*wPerf)/kCond))))/(4*pi*powf(rVar,3)*(-kCond*powf(mueff,2)+cblood*wPerf)*(exp(2*r2*sqrt((cblood*wPerf)/kCond))*(-1+r2*sqrt((cblood*wPerf)/kCond))+exp(2*r1*sqrt((cblood*wPerf)/kCond))*(1+r2*sqrt((cblood*wPerf)/kCond))))+(exp(-mueff*(r1+rVar)+(2*r1+rVar)*sqrt((cblood*wPerf)/kCond))*(-exp(mueff*r1+r2*sqrt((cblood*wPerf)/kCond))*powf(mueff,2)*Power*powf(r2,2)*(1+mueff*rVar)-exp(mueff*rVar+r1*sqrt((cblood*wPerf)/kCond))*powf(mueff,2)*Power*powf(rVar,2)*(1+r2*sqrt((cblood*wPerf)/kCond))-4*exp(mueff*(r1+rVar)+r1*sqrt((cblood*wPerf)/kCond))*pi*r1*powf(rVar,2)*(u0-ua)*(kCond*powf(mueff,2)-cblood*wPerf)*(1+r2*sqrt((cblood*wPerf)/kCond))))/(4*pi*powf(r1,2)*powf(rVar,3)*(-kCond*powf(mueff,2)+cblood*wPerf)*(exp(2*r2*sqrt((cblood*wPerf)/kCond))*(-1+r2*sqrt((cblood*wPerf)/kCond))+exp(2*r1*sqrt((cblood*wPerf)/kCond))*(1+r2*sqrt((cblood*wPerf)/kCond))));

//	*temperature = ua-(exp(-mueff*rVar)*mueff*mueff*Power)/(4*kCond*mueff*mueff*pi*rVar-4*cblood*pi*rVar*wPerf)+(exp(-mueff*(r1+rVar)+(r1+r2-rVar)*sqrt((cblood*wPerf)/kCond))*(exp(r1*(mueff+sqrt((cblood*wPerf)/kCond)))*mueff*mueff*Power*r2*r2*(1+mueff*rVar)-exp(mueff*rVar+r2*sqrt((cblood*wPerf)/kCond))*mueff*mueff*Power*rVar*rVar*(-1+r2*sqrt((cblood*wPerf)/kCond))+4*exp(mueff*(r1+rVar)+r2*sqrt((cblood*wPerf)/kCond))*pi*r1*rVar*rVar*(u0-ua)*(-kCond*mueff*mueff+cblood*wPerf)*(-1+r2*sqrt((cblood*wPerf)/kCond))))/(4*pi*rVar*rVar*rVar*(-kCond*mueff*mueff+cblood*wPerf)*(exp(2*r2*sqrt((cblood*wPerf)/kCond))*(-1+r2*sqrt((cblood*wPerf)/kCond))+exp(2*r1*sqrt((cblood*wPerf)/kCond))*(1+r2*sqrt((cblood*wPerf)/kCond))))+(exp(-mueff*(r1+rVar)+(2*r1+rVar)*sqrt((cblood*wPerf)/kCond))*(-exp(mueff*r1+r2*sqrt((cblood*wPerf)/kCond))*mueff*mueff*Power*r2*r2*(1+mueff*rVar)-exp(mueff*rVar+r1*sqrt((cblood*wPerf)/kCond))*mueff*mueff*Power*rVar*rVar*(1+r2*sqrt((cblood*wPerf)/kCond))-4*exp(mueff*(r1+rVar)+r1*sqrt((cblood*wPerf)/kCond))*pi*r1*rVar*rVar*(u0-ua)*(kCond*mueff*mueff-cblood*wPerf)*(1+r2*sqrt((cblood*wPerf)/kCond))))/(4*pi*r1*r1*rVar*rVar*rVar*(-kCond*mueff*mueff+cblood*wPerf)*(exp(2*r2*sqrt((cblood*wPerf)/kCond))*(-1+r2*sqrt((cblood*wPerf)/kCond))+exp(2*r1*sqrt((cblood*wPerf)/kCond))*(1+r2*sqrt((cblood*wPerf)/kCond))));

//   *temperature = ua+(P*PI_Var*(mueff*mueff)*exp(-mueff*r)*(1.0/4.0))/(r*(w-k*(mueff*mueff)))-(exp(-R1*mueff-R2*mueff)*exp(r*sqrt(w/k))*(P*PI_Var*(mueff*mueff)*exp(R1*sqrt(w/k))*exp(R2*mueff)-P*PI_Var*(mueff*mueff)*exp(R2*sqrt(w/k))*exp(R1*mueff)-P*PI_Var*R2*(mueff*mueff*mueff)*exp(R2*sqrt(w/k))*exp(R1*mueff)-R1*u0*w*exp(R1*sqrt(w/k))*exp(R1*mueff)*exp(R2*mueff)*4.0+R1*ua*w*exp(R1*sqrt(w/k))*exp(R1*mueff)*exp(R2*mueff)*4.0+R1*k*(mueff*mueff)*u0*exp(R1*sqrt(w/k))*exp(R1*mueff)*exp(R2*mueff)*4.0-R1*k*(mueff*mueff)*ua*exp(R1*sqrt(w/k))*exp(R1*mueff)*exp(R2*mueff)*4.0+P*PI_Var*R2*(mueff*mueff)*exp(R1*sqrt(w/k))*exp(R2*mueff)*sqrt(w/k)-R1*R2*u0*w*exp(R1*sqrt(w/k))*exp(R1*mueff)*exp(R2*mueff)*sqrt(w/k)*4.0+R1*R2*ua*w*exp(R1*sqrt(w/k))*exp(R1*mueff)*exp(R2*mueff)*sqrt(w/k)*4.0+R1*R2*k*(mueff*mueff)*u0*exp(R1*sqrt(w/k))*exp(R1*mueff)*exp(R2*mueff)*sqrt(w/k)*4.0-R1*R2*k*(mueff*mueff)*ua*exp(R1*sqrt(w/k))*exp(R1*mueff)*exp(R2*mueff)*sqrt(w/k)*4.0)*(1.0/4.0))/(r*(w-k*(mueff*mueff))*(exp(R1*sqrt(w/k)*2.0)-exp(R2*sqrt(w/k)*2.0)+R2*exp(R1*sqrt(w/k)*2.0)*sqrt(w/k)+R2*exp(R2*sqrt(w/k)*2.0)*sqrt(w/k)))-(exp(R1*sqrt(w/k))*exp(R2*sqrt(w/k))*exp(-r*sqrt(w/k))*exp(-R1*mueff)*exp(-R2*mueff)*(P*PI_Var*(mueff*mueff)*exp(R1*sqrt(w/k))*exp(R1*mueff)-P*PI_Var*(mueff*mueff)*exp(R2*sqrt(w/k))*exp(R2*mueff)+P*PI_Var*R2*(mueff*mueff*mueff)*exp(R1*sqrt(w/k))*exp(R1*mueff)+R1*u0*w*exp(R2*sqrt(w/k))*exp(R1*mueff)*exp(R2*mueff)*4.0-R1*ua*w*exp(R2*sqrt(w/k))*exp(R1*mueff)*exp(R2*mueff)*4.0-R1*k*(mueff*mueff)*u0*exp(R2*sqrt(w/k))*exp(R1*mueff)*exp(R2*mueff)*4.0+R1*k*(mueff*mueff)*ua*exp(R2*sqrt(w/k))*exp(R1*mueff)*exp(R2*mueff)*4.0+P*PI_Var*R2*(mueff*mueff)*exp(R2*sqrt(w/k))*exp(R2*mueff)*sqrt(w/k)-R1*R2*u0*w*exp(R2*sqrt(w/k))*exp(R1*mueff)*exp(R2*mueff)*sqrt(w/k)*4.0+R1*R2*ua*w*exp(R2*sqrt(w/k))*exp(R1*mueff)*exp(R2*mueff)*sqrt(w/k)*4.0+R1*R2*k*(mueff*mueff)*u0*exp(R2*sqrt(w/k))*exp(R1*mueff)*exp(R2*mueff)*sqrt(w/k)*4.0-R1*R2*k*(mueff*mueff)*ua*exp(R2*sqrt(w/k))*exp(R1*mueff)*exp(R2*mueff)*sqrt(w/k)*4.0)*(1.0/4.0))/(r*(w-k*(mueff*mueff))*(exp(R1*sqrt(w/k)*2.0)-exp(R2*sqrt(w/k)*2.0)+R2*exp(R1*sqrt(w/k)*2.0)*sqrt(w/k)+R2*exp(R2*sqrt(w/k)*2.0)*sqrt(w/k)));
}
__device__
void DebugWrite(int idx,int idmat,double rad,double omega, double conduction, double mueff,double temp)
{
   printf("%d %d %12.5e %12.5e %12.5e %12.5e %12.5e\n",idx,idmat,rad,omega,conduction,mueff,temp);
   //int j,k;

   //for (j=0;j<n;j++) {
   //   for (k=0;k<n+1;k++) {
   //      printf("%d %d %12.5e %12.5e ",k,j,a[k][j].real(),a[k][j].imag());
   //   }
   //   printf(" | %d  %12.5e %12.5e \n",j,x[j].real(),x[j].imag());
   //}
   //printf("\n");
}

/*
 * Device code
 */
__global__ 
void steadyStatePennesLaser(
         int const NTissue,
         const    int* MaterialID,
         const double* Perfusion,
         const double* ThermalConduction,
         const double* EffectiveAttenuation,
         double const innerRadius,
         double const outerRadius,
         int const NSource,
         double const Power,
         const double* SourceXloc,
         const double* SourceYloc,
         const double* SourceZloc,
         double const InitialTemperature,
         double const ArterialTemperature,
         double const SpecificHeatBlood,
	 double const SpacingX,
	 double const SpacingY,
	 double const SpacingZ,
         int const NpixelX,
         int const NpixelY,
         int const NpixelZ,
         double* d_TemperatureArray)
{

//     double SpacingX=0.00078;
    /*
      grid stride loop design pattern, 1-d grid
      http://devblogs.nvidia.com/parallelforall/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
         - By using a loop, you can support any problem size even if it exceeds the largest grid size your CUDA device supports. Moreover, you can limit the number of blocks you use to tune performance.
    */
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; 
         idx < NpixelX * NpixelY * NpixelZ;
         idx += blockDim.x * gridDim.x) 
      {
        // compute indices
        int index = idx; // use dummy variable
        int kkk = index/(NpixelX*NpixelY); 
        index -= kkk*NpixelX*NpixelY; 
        
        int jjj = index/NpixelX; 
        index -= jjj*NpixelX; 
        
        int iii = index/1;

        /* get material parameters */
        int const idmaterial =  MaterialID[idx];
        double omega      = Perfusion[idmaterial];
        double conduction = ThermalConduction[idmaterial];
        double mueff      = EffectiveAttenuation[idmaterial];
//	printf("%d",mueff);
        // linear superpostion of temperature sources
        double temperature = 0.0;
        for (int lll=0;lll<NSource;lll++) 
          {
//           double radiusSQ = (iii * SpacingX + 0.13281 - SourceXloc[lll])*(iii * SpacingX + 0.13281 - SourceXloc[lll])
//                           + (jjj * SpacingY + 0.10547 - SourceYloc[lll])*(jjj * SpacingY + 0.10547 - SourceYloc[lll])
//                           + (kkk * SpacingZ + 0.06000 - SourceZloc[lll])*(kkk * SpacingZ + 0.06000- SourceZloc[lll]);

	   double radiusSQ=powf(iii*SpacingX-SourceXloc[lll],2)
			  +powf(jjj*SpacingY-SourceYloc[lll],2)
			  +powf(kkk*SpacingZ-SourceZloc[lll],2);//SourceXloc[0]*SourceXloc[0];
           double radius   = sqrt(radiusSQ);

           // call GF code 
	   double sourcetemperature;
           pointSource(radius, innerRadius, outerRadius, omega , SpecificHeatBlood, conduction , mueff, InitialTemperature, ArterialTemperature, Power , &sourcetemperature);

	   if (radius <= innerRadius && NSource ==1)
		{
                    sourcetemperature = InitialTemperature;
		}
           if (radius <= innerRadius && NSource == 10)
		{
                    sourcetemperature = InitialTemperature+55;
		}
           if (radius <= innerRadius && NSource > 1)
		{
                   sourcetemperature = InitialTemperature;
		}
           // DebugWrite(idx,idmaterial,radius,omega,conduction,mueff,sourcetemperature);
           // superposition
	   if (idmaterial==0)
	 	{
		   temperature=0;
		}
	   else
		{
                   temperature = temperature + sourcetemperature/((double)NSource); 		
		}	 
          }
        // store temperature in array
        d_TemperatureArray[idx] = temperature;
      }
}


